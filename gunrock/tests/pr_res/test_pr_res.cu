#include "hip/hip_runtime.h"
#include "pr_res/pr_res_enactor.hxx"
#include "test_utils.hxx"

#include <algorithm>
#include <cstdlib>

using namespace gunrock;
using namespace gunrock::pr_res;

int main(int argc, char** argv) {

    // read in graph file
    std::string filename;
    CommandLineArgs args(argc, argv);
    args.GetCmdLineArgument("file", filename);

    // read in source node from cmd line
    int max_iter = 50;
    args.GetCmdLineArgument("max_iter", max_iter);

    float lambda = 0.85;
    args.GetCmdLineArgument("lambda", lambda);

    float epsilon = 0.01;
    args.GetCmdLineArgument("epsilon", epsilon);

    bool undirected = true;
    args.GetCmdLineArgument("undirected", undirected);

    int device = 1;
    args.GetCmdLineArgument("device", device);

    hipSetDevice(device);
    cout << "max iter: "<< max_iter << " lambda: "<< lambda << " epsilon: "<< epsilon << " undirected: "<< undirected << " device: "<< device <<endl;

    // CUDA context is used for all mgpu transforms
    standard_context_t context;
   
    // Load graph data to device
    std::shared_ptr<graph_t> graph = load_graph(filename.c_str(), undirected);
    std::shared_ptr<graph_device_t> d_graph(std::make_shared<graph_device_t>());
    graph_to_device(d_graph, graph, context);

//    display_csr(graph.get()->csr);

    // Initializes coloring problem object
    std::shared_ptr<pr_res_problem_t> pr_res_problem(std::make_shared<pr_res_problem_t>(d_graph, max_iter, lambda, epsilon, context));
    cout << pr_res_problem.get()->gslice->num_nodes << ", "<< pr_res_problem.get()->gslice->num_edges << endl;


    std::shared_ptr<pr_res_enactor_t> pr_res_enactor(std::make_shared<pr_res_enactor_t>(context, d_graph->num_nodes, d_graph->num_edges));
    std::cout << "start PageRank" << std::endl;

    test_timer_t timer;
    timer.start();
    pr_res_enactor->enact(pr_res_problem, context);
    cout << "elapsed time: " << timer.end() << "s." << std::endl;

//    cout << "rank\n";
//    display_device_data(pr_res_problem.get()->d_rank.data(), pr_res_problem.get()->gslice->num_nodes);
//    cout << "res1\n";
//    display_device_data(pr_res_problem.get()->d_res1.data(), pr_res_problem.get()->gslice->num_nodes);


    pr_res_problem->extract(context);

    std::vector<float> validation_rank;
    std::vector<float> validation_res;

    pr_res_problem->cpu(validation_rank, validation_res, graph->csr->offsets, graph->csr->indices);

//    if (!validate(pr_res_problem.get()->h_rank, validation_rank, pr_res_problem.get()->epsilon*2))
    if (!validate_rank(pr_res_problem.get()->h_rank, validation_rank))
        cout << "Validation Error." << endl;
    else
        cout << "Correct." << endl;


}



